#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "graph-cuda.h"

__global__ 
void findPath(int *E, int *V, int *F, int *visited, int *parentP, int *flag,
    int *G, int num_vertices) {

    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id >= num_vertices) {
        // *flag = 0;
        return;
    }

    // printf("id = %d F[%d] = %d\n", id, id, F[id]);
    if (F[id] != 0 && visited[id] == 0) {
        F[id] = 0;
        visited[id] = 1;
        __syncthreads(); 
        int start = V[id];
        int end = V[id+1];
        // printf("%d to %d\n", start, end);
        for (int i = start; i < end; i++) {
            int nid = E[i];
            // printf("edge from %d to %d\n", id, nid);
            // printf("weight = %d\n", G[(id * num_vertices) + nid]);
            if (visited[nid] == 0 && G[(id * num_vertices) + nid] > 0) {
                parentP[nid] = id;
                F[nid] = 1;
                *flag = 0;
            }
        }
    }
}


long Graph::maxFlow(int s, int t) {
    long max_flow = 0;

    const size_t VERTEX_ROW_SIZE = sizeof(int) * num_vertices;

    int *parentP_host = (int *)malloc(VERTEX_ROW_SIZE);
    int *parentP_cuda;
    hipMalloc(&parentP_cuda, VERTEX_ROW_SIZE);
    hipMemset(parentP_cuda, 0, VERTEX_ROW_SIZE);

    int *graph_cuda;
    hipMalloc(&graph_cuda, VERTEX_ROW_SIZE * num_vertices);
    for (int row = 0; row < num_vertices; row++)
        hipMemcpy(&graph_cuda[row * num_vertices], graph[row].data(), 
            VERTEX_ROW_SIZE, hipMemcpyHostToDevice);

    int *E;
    hipMalloc(&E, sizeof(int) * num_edges);
    hipMemcpy(E, edges.data(), sizeof(int) * num_edges, hipMemcpyHostToDevice);

    int *V;
    hipMalloc(&V, VERTEX_ROW_SIZE + sizeof(int));
    hipMemcpy(V, nodes.data(), VERTEX_ROW_SIZE + sizeof(int), hipMemcpyHostToDevice);

    int *F_temp = (int *)calloc(num_vertices, sizeof(int));
    F_temp[s] = 1;
    int *F;
    hipMalloc(&F, VERTEX_ROW_SIZE);
    hipMemcpy(F, F_temp, VERTEX_ROW_SIZE, hipMemcpyHostToDevice);
    
    int *visited_host = (int *)malloc(VERTEX_ROW_SIZE);
    int *visited_cuda;
    hipMalloc(&visited_cuda, VERTEX_ROW_SIZE);
    hipMemset(visited_cuda, 0, VERTEX_ROW_SIZE);

    int *flag_host = (int *)malloc(sizeof(int));
    int *flag_cuda;
    hipMalloc(&flag_cuda, sizeof(int));
    hipMemset(flag_cuda, 0, sizeof(int));

    int threadsPerBlock = 32;
    int blocks = 1 + (num_vertices / threadsPerBlock);

    *flag_host = 0;
    while (*flag_host == 0) {
        *flag_host = 1;
        hipMemcpy(flag_cuda, flag_host, sizeof(int), hipMemcpyHostToDevice);
        findPath<<<blocks, threadsPerBlock >>>(E, V, F, visited_cuda,
            parentP_cuda, flag_cuda, graph_cuda, num_vertices);
        hipMemcpy(flag_host, flag_cuda, sizeof(int), hipMemcpyDeviceToHost);
    }
    hipMemcpy(parentP_host, parentP_cuda, num_vertices * sizeof(int),
        hipMemcpyDeviceToHost);
    hipMemcpy(visited_host, visited_cuda, num_vertices * sizeof(int),
        hipMemcpyDeviceToHost);
    int isPath = (visited_host[t] != 0);

    while (isPath) {
        long flow = LONG_MAX;
        for (int v = t; v != s; v = parentP_host[v]){
            int u = parentP_host[v];
            flow = std::min(flow, (long) graph[u][v]);
        }
        for (int v = t; v != s; v = parentP_host[v]) {
            int u = parentP_host[v];
            graph[u][v] -= flow;
            graph[v][u] += flow;
        }
        max_flow += flow;

        hipMemset(parentP_cuda, 0, VERTEX_ROW_SIZE);
        for (int row = 0; row < num_vertices; row++)
            hipMemcpy(&graph_cuda[row * num_vertices], graph[row].data(), 
                VERTEX_ROW_SIZE, hipMemcpyHostToDevice);
        hipMemcpy(F, F_temp, VERTEX_ROW_SIZE, hipMemcpyHostToDevice);
        hipMemset(visited_cuda, 0, VERTEX_ROW_SIZE);

        *flag_host = 0;
        while (*flag_host == 0) {
            *flag_host = 1;
            hipMemcpy(flag_cuda, flag_host, sizeof(int), hipMemcpyHostToDevice);
            findPath<<<blocks, threadsPerBlock >>>(E, V, F, visited_cuda,
                parentP_cuda, flag_cuda, graph_cuda, num_vertices);
            hipMemcpy(flag_host, flag_cuda, sizeof(int), hipMemcpyDeviceToHost);
        }
        hipMemcpy(parentP_host, parentP_cuda, num_vertices * sizeof(int),
            hipMemcpyDeviceToHost);
        hipMemcpy(visited_host, visited_cuda, num_vertices * sizeof(int),
            hipMemcpyDeviceToHost);
        isPath = (visited_host[t] != 0);
    }
    return max_flow;
}