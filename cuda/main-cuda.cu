#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "CycleTimer.h"
#include "graph-cuda.h"

__global__ 
void findPath(int* E, int* V, int* F, int* visited, int* parentP, int* flag, int* G, int* num_vertices){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > (*num_vertices)){
        *flag = 0;
        return;
    }

    printf("id = %d F[%d] = %d\n",id,id,F[id]);

    if (F[id] == 1 && visited[id] == 0)
    {
        F[id] = 0;
        visited[id] = 1;
        __syncthreads(); 
        int start = V[id];
        int end = V[id+1];
        printf("%d to %d\n", start, end);
        for (int i = start; i < end; i++) 
        {
            int nid = E[i];
            printf("edge from %d to %d\n", id, nid);
            printf("weight = %d\n", G[(id * (*num_vertices)) + nid]);
            if (visited[nid] == 0 && (G[(id * (*num_vertices)) + nid] > 0))
            {
                parentP[nid] = id;
                F[nid] = 1;
                *flag = 0;
            }
        }
    }
}


long Graph::maxFlow(int s, int t){
    long max_flow = 0;

    int *parentP_cuda;
    int *parentP_host = (int *)calloc(num_vertices, sizeof(int));
    hipMalloc(&parentP_cuda, sizeof(int) * num_vertices);
    hipMemset(parentP_cuda, 0, sizeof(int) * num_vertices);

    int* graph_cuda;
    hipMalloc(&graph_cuda, sizeof(int) * num_vertices * num_vertices);
    for (int row = 0; row < num_vertices; row++)
        hipMemcpy(&(graph_cuda[row * num_vertices]), graph[row].data(), 
            num_vertices, hipMemcpyHostToDevice);

    int* E;
    hipMalloc(&E, sizeof(int) * num_edges);
    hipMemcpy(E, edges.data(), sizeof(int) * num_edges,
        hipMemcpyHostToDevice);

    int* V;
    hipMalloc(&V, sizeof(int)*(num_vertices + 1));
    hipMemcpy(V, nodes.data(), sizeof(int) * (num_vertices + 1),
        hipMemcpyHostToDevice);

    int* F;
    hipMalloc(&F, sizeof(int) * num_vertices);
    hipMemset(F, 0, sizeof(int) * num_vertices);
    hipMemset(&(F[s]), 1 , sizeof(int));
    
    int* visited_cuda;
    int* visited_host = (int *)calloc(num_vertices, sizeof(int));
    hipMalloc(&visited_cuda, sizeof(int) * num_vertices);
    hipMemset(visited_cuda, 0, sizeof(int) * num_vertices);

    int *flag_cuda;
    int *flag_host;
    flag_host = (int *)calloc(1, sizeof(int));
    hipMalloc(&flag_cuda, sizeof(int));
    hipMemset(flag_cuda, 0, sizeof(int));

    int* nv;
    hipMalloc(&nv, sizeof(int));
    hipMemcpy(nv, &num_vertices, sizeof(int), hipMemcpyHostToDevice);

    int blocks = 1;
    int threadsPerBlock = 32;

    int isPath = 0;

    *flag_host = 0;
    while (*flag_host == 0){
        *flag_host = 1;
        hipMemcpy(flag_cuda, flag_host, sizeof(int), hipMemcpyHostToDevice);
        findPath <<<blocks, threadsPerBlock >>>(E, V, F, visited_cuda,
            parentP_cuda, flag_cuda, graph_cuda, nv);
        // for (int x = 0; x < num_vertices; x++){
        //     printf("%d\n", visited_host[x]);
        // }
        hipMemcpy(flag_host, flag_cuda, sizeof(int), hipMemcpyDeviceToHost);
    }

    hipMemcpy(parentP_host, parentP_cuda, num_vertices * sizeof(int),
        hipMemcpyDeviceToHost);
    hipMemcpy(visited_host, visited_cuda, num_vertices * sizeof(int),
        hipMemcpyDeviceToHost);

    

    isPath = (visited_host[t] == 1);

    while (isPath){
        long flow = LONG_MAX;
        for (int v = t; v != s; v = parentP_host[v]){
            int u = parentP_host[v];
            flow = std::min(flow, (long) graph[u][v]);
        }
        for (int v = t; v != s; v = parentP_host[v]){
            int u = parentP_host[v];
            graph[u][v] -= flow;
            graph[v][u] += flow;
        }
        max_flow += flow;

        hipMemset(parentP_cuda, 0, sizeof(int) * num_vertices);
        for (int row = 0; row < num_vertices; row++)
            hipMemcpy(&(graph_cuda[row * num_vertices]), graph[row].data(), 
                num_vertices, hipMemcpyHostToDevice);
        hipMemset(F, 0, num_vertices);
        hipMemset(&F[s], 1 , sizeof(int));
        hipMemset(&visited_cuda, 0, num_vertices * sizeof(int));

        *flag_host = 0;
        while (*flag_host == 0){
            *flag_host = 1;
            hipMemcpy(&flag_cuda, &flag_host, sizeof(int),
                hipMemcpyHostToDevice);
            findPath <<<blocks, threadsPerBlock >>>(E, V, F, visited_cuda,
                parentP_cuda, flag_cuda, graph_cuda, nv);
            hipMemcpy(&flag_host, &flag_cuda , sizeof(int), hipMemcpyDeviceToHost);
        }

        hipMemcpy(parentP_host, parentP_cuda, num_vertices * sizeof(int),
            hipMemcpyDeviceToHost);
        hipMemcpy(visited_host, visited_cuda, num_vertices * sizeof(int),
            hipMemcpyDeviceToHost);

        isPath = (visited_host[t] == 1);
    }
    return max_flow;
}